#include "hip/hip_runtime.h"
/*
 * Copyright 2016
 *
 * OfflineLoader.cu
 *
 *  Created on: 14.06.2016
 *      Author: Tobias Frust (t.frust@hzdr.de)
 */

#include <risa/ConfigReader/ConfigReader.h>
#include <risa/Loader/OfflineLoader.h>
#include <risa/Basics/performance.h>

#include <ddrf/MemoryPool.h>

#include <exception>
#include <fstream>
#include <chrono>

namespace risa {

OfflineLoader::OfflineLoader(const std::string& address,
      const std::string& configFile) :
      worstCaseTime_ { 0.0 }, bestCaseTime_ { std::numeric_limits<double>::max() } {
   if (readConfig(configFile)) {
      throw std::runtime_error(
            "recoLib::OfflineLoader: Configuration file could not be loaded successfully. Please check!");
   }

   stopFrame_ = 100000u;
   index_ = 1000u;

   memoryPoolIndex_ = ddrf::MemoryPool<manager_type>::instance()->registerStage(
         (numberOfFrames_ + 1) * numberOfPlanes_,
         numberOfProjections_ * numberOfDetectors_);

   readInput();
}

OfflineLoader::~OfflineLoader() {
   ddrf::MemoryPool<manager_type>::instance()->freeMemory(memoryPoolIndex_);
   BOOST_LOG_TRIVIAL(info)<< "recoLib::OfflineLoader: WorstCaseTime: " << worstCaseTime_ << "s; BestCaseTime: " << bestCaseTime_ << "s;";
}

auto OfflineLoader::loadImage() -> ddrf::Image<manager_type> {
   if (buffer_.empty())
      return ddrf::Image<manager_type>();
   auto sino = std::move(buffer_.front());
   if (sino.index() > 0) {
      tmr_.stop();
      double duration = tmr_.elapsed();
      if (duration < bestCaseTime_)
         bestCaseTime_ = duration;
      if (duration > worstCaseTime_)
         worstCaseTime_ = duration;
   }
   buffer_.pop();
   if (index_ < stopFrame_) {
      auto img = ddrf::MemoryPool<manager_type>::instance()->requestMemory(
            memoryPoolIndex_);
      img.setIdx(index_);
      buffer_.push(std::move(img));
      index_++;
   }
   tmr_.start();
   sino.setStart(std::chrono::high_resolution_clock::now());
   return sino;
}

auto OfflineLoader::readInput() -> void {
   Timer tmr1, tmr2;
   std::vector<std::vector<unsigned short>> fileContents(
         numberOfDetectorModules_);
   int numberOfDetPerModule = numberOfDetectors_ / numberOfDetectorModules_;
   if (path_.back() != '/')
      path_.append("/");
   tmr1.start();
   tmr2.start();
#pragma omp parallel for default(shared) num_threads(numberOfDetectorModules_/3)
   for (auto i = 1; i <= numberOfDetectorModules_; i++) {
      std::vector<unsigned short> content;
      std::ifstream input(path_ + fileName_ + std::to_string(i) + fileEnding_,
            std::ios::in | std::ios::binary);
      if (!input) {
         BOOST_LOG_TRIVIAL(error)<< "recoLib::OfflineLoader: Source file could not be loaded.";
         throw std::runtime_error("File could not be opened. Please check!");
      }
      //allocate memory in vector
      std::streampos fileSize;
      input.seekg(0, std::ios::end);
      fileSize = input.tellg();
      input.seekg(0, std::ios::beg);
      content.resize(fileSize / sizeof(unsigned short));
      input.read((char*) &content[0], fileSize);
      fileContents[i - 1] = content;
   }
   tmr2.stop();
   for (unsigned int i = 0; i < numberOfFrames_; i++) {
      for (auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++) {
         auto sino = ddrf::MemoryPool<manager_type>::instance()->requestMemory(memoryPoolIndex_);
//            for (auto projInd = 0; projInd < numberOfProjections_; projInd++) {
//               for (auto detModInd = 0; detModInd < numberOfDetectorModules_;
//                     detModInd++) {
//                  unsigned int startIndex = projInd * numberOfDetPerModule
//                        + (planeInd + i * numberOfPlanes_) * numberOfDetPerModule * numberOfProjections_;
//                  unsigned int indexSorted = detModInd * numberOfDetPerModule
//                        + projInd * numberOfDetectors_;
//                  std::copy(fileContents[detModInd].begin() + startIndex,
//                        fileContents[detModInd].begin() + startIndex
//                              + numberOfDetPerModule,
//                        sino.container().get() + indexSorted);
//               }
//            }
//            sino.setIdx(planeInd + i * numberOfPlanes_);
//            sino.setPlane(planeInd);
//            buffer_.push(std::move(sino));
//         }
         for (auto detModInd = 0; detModInd < numberOfDetectorModules_;
               detModInd++) {
            std::size_t startIndex = (planeInd + i * numberOfPlanes_) * numberOfDetPerModule * numberOfProjections_;
            std::copy(fileContents[detModInd].cbegin() + startIndex, fileContents[detModInd].cbegin() + startIndex
                        + numberOfDetPerModule * numberOfProjections_, sino.container().get()
                        + detModInd * numberOfDetPerModule * numberOfProjections_);
         }
         sino.setIdx(planeInd + i * numberOfPlanes_);
         sino.setPlane(planeInd);
         buffer_.push(std::move(sino));
      }
   }
   tmr1.stop();
   double totalFileSize = numberOfProjections_ * numberOfDetectors_
         * numberOfPlanes_ * numberOfFrames_ * sizeof(unsigned short) / 1024.0
         / 1024.0;
   BOOST_LOG_TRIVIAL(info)<< "recoLib::OfllineLoader: Reading and sorting data input took " << tmr1.elapsed() << " s, " << totalFileSize/tmr2.elapsed() << " MByte/s.";
}

/**
 * All values needed for setting up the class are read from the config file
 * in this function.
 *
 * @param[in] configFile path to config file
 *
 * @return returns true, if configuration file could be read successfully, else false
 */
auto OfflineLoader::readConfig(const std::string& configFile) -> bool {
   ConfigReader configReader = ConfigReader(configFile.data());
   int samplingRate, scanRate;
   if (configReader.lookupValue("numberOfFanDetectors", numberOfDetectors_)
         && configReader.lookupValue("numberOfDetectorModules",
               numberOfDetectorModules_)
         && configReader.lookupValue("dataInputPath", path_)
         && configReader.lookupValue("dataFileName", fileName_)
         && configReader.lookupValue("dataFileEnding", fileEnding_)
         && configReader.lookupValue("numberOfPlanes", numberOfPlanes_)
         && configReader.lookupValue("samplingRate", samplingRate)
         && configReader.lookupValue("scanRate", scanRate)
         && configReader.lookupValue("numberOfDataFrames", numberOfFrames_)) {
      numberOfProjections_ = samplingRate * 1000000 / scanRate;
      return EXIT_SUCCESS;
   }
   return EXIT_FAILURE;
}

}
