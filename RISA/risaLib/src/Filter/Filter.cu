#include "hip/hip_runtime.h"
/*
 *  Copyright 2016
 *
 *  Filter.cu
 *
 *  Created on: 28.04.2016
 *      Author: Tobias Frust (t.frust@hzdr.de)
 *
 */

#include <risa/Filter/Filter.h>
#include <risa/ConfigReader/ConfigReader.h>
#include <risa/Basics/performance.h>
#include "cuda_kernels_filter.h"

#include <ddrf/cuda/Launch.h>
#include <ddrf/cuda/Check.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

__constant__ float filter_d[2049];

__global__ void applyFilter(const int x, const int y, const float normalization, hipfftComplex *data);

Filter::Filter(const std::string& configFile) {

   if (readConfig(configFile)) {
      throw std::runtime_error(
            "recoLib::cuda::Filter: Configuration file could not be loaded successfully. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //cuFFT library is initialized for each device
   for (auto i = 0; i < numberOfDevices_; i++) {
      initCuFFT(i);
   }

   designFilter();

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Filter::processor, this, i };
   }
   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Filter: Running " << numberOfDevices_ << " Threads.";
}

Filter::~Filter() {
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
      CHECK_CUFFT(hipfftDestroy(plansFwd_[i]));
      CHECK_CUFFT(hipfftDestroy(plansInv_[i]));
   }
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Filter: Destroyed.";
}

auto Filter::process(input_type&& sinogram) -> void {
   if (sinogram.valid()) {
      BOOST_LOG_TRIVIAL(debug) << "Filter: Image arrived with Index: " << sinogram.index() << "to device " << sinogram.device();
      sinograms_[sinogram.device()].push(std::move(sinogram));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Filter: Received sentinel, finishing.";

      //send sentinal to processor thread and wait 'til it's finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         sinograms_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }
      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::Filter: Finished.";
   }
}

auto Filter::wait() -> output_type {
   return results_.take();
}

/**
 * The processor()-Method takes one sinogram from the queue. Via the cuFFT-Library
 * it is transformed into frequency space for applying the filter function.
 * After filtering the transformation is reverted via the inverse fourier transform.
 * Finally, the filtered sinogram is pushed back into the output queue for
 * further processing.
 *
 */
auto Filter::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "Filter");
   CHECK(hipSetDevice(deviceID));
   auto sinoFreq = ddrf::cuda::make_device_ptr<hipfftComplex,
         ddrf::cuda::async_copy_policy>(
         numberOfProjections_ * ((numberOfDetectors_ / 2.0) + 1));
   dim3 dimBlock(blockSize2D_, blockSize2D_);
   dim3 dimGrid((int) ceil((numberOfDetectors_ / 2.0 + 1) / (float) blockSize2D_),
         (int) ceil(numberOfProjections_ / (float) blockSize2D_));
   CHECK(
         hipMemcpyToSymbol(HIP_SYMBOL(filter_d), filter_.data(), sizeof(float) * filter_.size()));
   const float normalizationFactor = 1.0/(float)numberOfDetectors_;
   BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::Filter: Running Thread for Device " << deviceID;
   while (true) {
      auto sinogram = sinograms_[deviceID].take();
      if (!sinogram.valid())
         break;
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Filter: Filtering sinogram with Index " << sinogram.index();

      //forward transformation
      CHECK_CUFFT(
            hipfftExecR2C(plansFwd_[deviceID],
                  (hipfftReal* ) sinogram.container().get(),
                  thrust::raw_pointer_cast(&(sinoFreq[0]))));

      //Filtering
      applyFilter<<<dimGrid, dimBlock, 0, streams_[deviceID]>>>(
            (numberOfDetectors_ / 2) + 1, numberOfProjections_, normalizationFactor,
            thrust::raw_pointer_cast(&(sinoFreq[0])));

//      filterRamp<<<dimGrid, dimBlock, 0, streams_[deviceID]>>>(
//            (numberOfDetectors_ / 2) + 1, numberOfProjections_,
//            thrust::raw_pointer_cast(&(sinoFreq[0])));
      CHECK(hipPeekAtLastError());

      //reverse transformation
      CHECK_CUFFT(
            hipfftExecC2R(plansInv_[deviceID],
                  thrust::raw_pointer_cast(sinoFreq.get()),
                  (hipfftReal* ) sinogram.container().get()));
      //wait until work on device is finished
      CHECK(hipStreamSynchronize(streams_[deviceID]));
      results_.push(std::move(sinogram));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Filter: Filtering sinogram with Index " << sinogram.index() << " finished.";
   }
}

/**
 * Initializes the NVIDIA cuFFT Library for use in processor-Method
 * Creates a stream in which all operations in this class are executed.
 * This is necessary, because the cuFFT library ignores the compiler
 * option 'default-stream per-thread'.
 * Finally, memory for the transformed sinogram is allocated here.
 *
 */
auto Filter::initCuFFT(const int deviceID) -> void {

   CHECK(hipSetDevice(deviceID));

   hipStream_t stream;
   hipfftHandle planFwd, planInv;

   CHECK(hipStreamCreate(&stream));
   streams_[deviceID] = stream;

   CHECK_CUFFT(
         hipfftPlanMany(&planFwd, 1, &numberOfDetectors_, NULL, 0, 0, NULL, 0, 0, HIPFFT_R2C, numberOfProjections_));

   CHECK_CUFFT(hipfftSetStream(planFwd, stream));

   CHECK_CUFFT(
         hipfftPlanMany(&planInv, 1, &numberOfDetectors_, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2R, numberOfProjections_));

   CHECK_CUFFT(hipfftSetStream(planInv, stream));

   plansFwd_[deviceID] = planFwd;
   plansInv_[deviceID] = planInv;
}

auto Filter::designFilter() -> void {
   int filterSize = numberOfDetectors_/2 + 1;
   filter_.reserve(filterSize);
   filter_.push_back(0.0);
   for(auto i = 1; i < filterSize; i++){
      //actual w at frequency axis
      const float w = 2 * M_PI * i / (float)numberOfDetectors_;
      if(w > M_PI*cutoffFraction_){
         filter_.push_back(0.0);
         continue;
      }
      float filterValue = 2 * i / (float)numberOfDetectors_; //* hanning(w, (float)1.0);
      if(filterType_ == detail::FilterType::hamming)
         filterValue *= hamming(w, cutoffFraction_);
      else if(filterType_ == detail::FilterType::hanning)
         filterValue *= hanning(w, cutoffFraction_);
      else if(filterType_ == detail::FilterType::sheppLogan)
         filterValue *= sheppLogan(w, cutoffFraction_);
      else if(filterType_ == detail::FilterType::cosine)
         filterValue *= cosine(w, cutoffFraction_);
      filter_.push_back(filterValue);
   }
}

/**
 * All values needed for setting up the class are read from the config file
 * in this function.
 *
 * @param[in] configFile path to config file
 *
 * @return returns true, if configuration file could be read successfully, else false
 */
auto Filter::readConfig(const std::string& configFile) -> bool {
   recoLib::ConfigReader configReader = recoLib::ConfigReader(
         configFile.data());
   std::string filterType;
   if (configReader.lookupValue("numberOfParallelProjections", numberOfProjections_)
         && configReader.lookupValue("numberOfParallelDetectors", numberOfDetectors_)
         && configReader.lookupValue("numberOfPixels", numberOfPixels_)
         && configReader.lookupValue("blockSize2D_filter", blockSize2D_)
         && configReader.lookupValue("filterType", filterType)
         && configReader.lookupValue("cutoffFraction", cutoffFraction_)){
      if(filterType == "ramp")
         filterType_ = detail::FilterType::ramp;
      else if(filterType == "sheppLogan")
         filterType_ = detail::FilterType::sheppLogan;
      else if(filterType == "hamming")
         filterType_ = detail::FilterType::hamming;
      else if(filterType == "hanning")
         filterType_ = detail::FilterType::hanning;
      else if(filterType == "cosine")
         filterType_ = detail::FilterType::cosine;
      else{
         BOOST_LOG_TRIVIAL(error) << "recoLib::cuda::Filter: Requested filter mode not supported. Using Ramp-Filter.";
         filterType_ = detail::FilterType::ramp;
      }
      return EXIT_SUCCESS;
   }
   return EXIT_FAILURE;
}

__global__ void applyFilter(const int x, const int y, const float normalization, hipfftComplex *data) {
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < x && j < y) {
      //cufft performs an unnormalized transformation ifft(fft(A))=length(A)*A
      //->normalization needs to be performed
      const float filterVal = filter_d[i] * normalization;
      data[i + j * x].x *= filterVal;
      data[i + j * x].y *= filterVal;
   }
}

}
}
