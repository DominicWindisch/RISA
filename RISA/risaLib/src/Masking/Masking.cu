#include "hip/hip_runtime.h"
/*
 * Copyright 2016
 *
 * CropImage.cu
 *
 *  Created on: 31.05.2016
 *      Author: Tobias Frust (t.frust@hzdr.de)
 */

#include <risa/Masking/Masking.h>
#include <risa/ConfigReader/ConfigReader.h>
#include <risa/Basics/performance.h>

#include <ddrf/cuda/Launch.h>
#include <ddrf/cuda/Check.h>
#include <ddrf/cuda/Coordinates.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

//!   This CUDA kernel multiplies the mask and the reconstructed image
/**
 * @param[in,out] img            the reconstructed image, that is multiplied with the mask in-place
 * @param[in]     value          the value, the pixels shall be replaced with
 * @param[in]     numberOfPixels the number of pixels in the reconstruction grid in one dimension
 */
__global__ void mask(float* __restrict__ img, const float value, const int numberOfPixels);

Masking::Masking(const std::string& configFile) {

   if (readConfig(configFile)) {
      throw std::runtime_error(
            "recoLib::cuda::CropImage: Configuration file could not be loaded successfully. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      hipStream_t stream;
      CHECK(hipStreamCreate(&stream));
      streams_[i] = stream;
   }

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Masking::processor, this, i };
   }
   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: Running " << numberOfDevices_ << " Threads.";
}

Masking::~Masking() {
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::CropImage: Destroyed.";
}

auto Masking::process(input_type&& img) -> void {
   if (img.valid()) {
      BOOST_LOG_TRIVIAL(debug)<< "CropImage: Image arrived with Index: " << img.index() << "to device " << img.device();
      imgs_[img.device()].push(std::move(img));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: Received sentinel, finishing.";

      //send sentinal to processor thread and wait 'til it's finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         imgs_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }
      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::CropImage: Finished.";
   }
}

auto Masking::wait() -> output_type {
   return results_.take();
}

auto Masking::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "CropImage");
   CHECK(hipSetDevice(deviceID));
   dim3 blocks(16, 16);
   dim3 grids(std::ceil(numberOfPixels_ / 16.0),
         std::ceil(numberOfPixels_ / 16.0));
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::CropImage: Running Thread for Device " << deviceID;
   while (true) {
      auto img = imgs_[deviceID].take();
      if (!img.valid())
         break;
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: CropImageing image with Index " << img.index();

      mask<<<grids, blocks, 0, streams_[deviceID]>>>(img.container().get(),
            0.0 ,numberOfPixels_);
      CHECK(hipPeekAtLastError());

      //wait until work on device is finished
      CHECK(hipStreamSynchronize(streams_[deviceID]));
      results_.push(std::move(img));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: CropImageing image with Index " << img.index() << " finished.";
   }
}

auto Masking::readConfig(const std::string& configFile) -> bool {
   ConfigReader configReader = ConfigReader(
         configFile.data());
   if (configReader.lookupValue("numberOfPixels", numberOfPixels_))
      return EXIT_SUCCESS;

   return EXIT_FAILURE;
}

__global__ void mask(float* __restrict__ img, const float value, const int numberOfPixels) {
   const auto x = ddrf::cuda::getX();
   const auto y = ddrf::cuda::getY();
   if (x >= numberOfPixels || y >= numberOfPixels)
      return;
   const float center = (numberOfPixels - 1.0) * 0.5;
   const float dX = x - center;
   const float dY = y - center;
   const float distance = dX * dX + dY * dY;
   if (distance > numberOfPixels * numberOfPixels * 0.25) {
      img[x + numberOfPixels * y] = value;
   }
}

}
}

