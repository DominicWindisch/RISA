#include "hip/hip_runtime.h"
/*
 * Copyright 2016
 *
 * Attenuation.cu
 *
 *  Created on: 02.06.2016
 *      Author: Tobias Frust (t.frust@hzdr.de)
 */

#include <risa/Attenuation/Attenuation.h>
#include <risa/ConfigReader/ConfigReader.h>
#include <risa/Basics/performance.h>

#include <ddrf/cuda/Launch.h>
#include <ddrf/cuda/Check.h>
#include <ddrf/cuda/Coordinates.h>
#include <ddrf/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <omp.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <iterator>
#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

Attenuation::Attenuation(const std::string& configFile) {

   if (readConfig(configFile)) {
      throw std::runtime_error(
            "recoLib::cuda::Attenuation: Configuration file could not be loaded successfully. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      memoryPoolIdxs_[i] =
            ddrf::MemoryPool<deviceManagerType>::instance()->registerStage(memPoolSize_,
                  numberOfDetectors_ * numberOfProjections_);
      hipStream_t stream;
      CHECK(hipStreamCreate(&stream));
      streams_[i] = stream;
   }

   init();

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Attenuation::processor, this, i };
   }

   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Running " << numberOfDevices_ << " Threads.";
}

Attenuation::~Attenuation() {
   for (auto idx : memoryPoolIdxs_) {
      CHECK(hipSetDevice(idx.first));
      ddrf::MemoryPool<deviceManagerType>::instance()->freeMemory(idx.second);
   }
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Destroyed.";
}

auto Attenuation::process(input_type&& sinogram) -> void {
   if (sinogram.valid()) {
      BOOST_LOG_TRIVIAL(debug)<< "Attenuation: Image arrived with Index: " << sinogram.index() << "to device " << sinogram.device();
      sinograms_[sinogram.device()].push(std::move(sinogram));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Received sentinel, finishing.";

      //send sentinal to processor thread and wait 'til it's finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         sinograms_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }
      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::Attenuation: Finished.";
   }
}

auto Attenuation::wait() -> output_type {
   return results_.take();
}

auto Attenuation::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "Attenuation");
   CHECK(hipSetDevice(deviceID));
   auto avgDark_d = ddrf::cuda::make_device_ptr<float>(avgDark_.size());
   auto avgReference_d = ddrf::cuda::make_device_ptr<float>(
         avgReference_.size());
   auto mask_d = ddrf::cuda::make_device_ptr<float>(
         numberOfDetectors_ * numberOfProjections_);
   CHECK(
         hipMemcpyAsync(avgDark_d.get(), avgDark_.data(),
               sizeof(float) * avgDark_.size(), hipMemcpyHostToDevice,
               streams_[deviceID]));
   CHECK(
         hipMemcpyAsync(avgReference_d.get(), avgReference_.data(),
               sizeof(float) * avgReference_.size(), hipMemcpyHostToDevice,
               streams_[deviceID]));
   //compute mask for relevant area
   std::vector<float> mask;
   relevantAreaMask(mask);
   CHECK(
         hipMemcpyAsync(mask_d.get(), mask.data(), sizeof(float) * mask.size(),
               hipMemcpyHostToDevice, streams_[deviceID]));

   dim3 blocks(blockSize2D_, blockSize2D_);
   dim3 grids(std::ceil(numberOfDetectors_ / (float)blockSize2D_),
         std::ceil(numberOfProjections_ / (float)blockSize2D_));
   float temp = pow(10, -5);
   CHECK(hipStreamSynchronize(streams_[deviceID]));
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Running Thread for Device " << deviceID;

   while (true) {
      auto sinogram = sinograms_[deviceID].take();
      if (!sinogram.valid())
         break;
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Attenuationing image with Index " << sinogram.index();

      auto sino =
            ddrf::MemoryPool<deviceManagerType>::instance()->requestMemory(
                  memoryPoolIdxs_[deviceID]);

      computeAttenuation<<<grids, blocks, 0, streams_[deviceID]>>>(
            sinogram.container().get(), mask_d.get(), sino.container().get(),
            avgReference_d.get(), avgDark_d.get(), temp, numberOfDetectors_,
            numberOfProjections_, sinogram.plane());
      CHECK(hipPeekAtLastError());

      sino.setIdx(sinogram.index());
      sino.setDevice(deviceID);
      sino.setPlane(sinogram.plane());

      //wait until work on device is finished
      CHECK(hipStreamSynchronize(streams_[deviceID]));
      results_.push(std::move(sino));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Attenuationing image with Index " << sinogram.index() << " finished.";
   }
}

auto Attenuation::init() -> void {
   std::vector<double> darkAverageDouble;
   readDarkInputFiles<double>(pathDark_, darkAverageDouble);

   avgDark_.resize(darkAverageDouble.size());

   //conversion from double to float
   std::copy(darkAverageDouble.begin(), darkAverageDouble.end(),
         avgDark_.begin());

//   //normalize the dark values by the number of reference frames
   std::transform(avgDark_.begin(), avgDark_.end(), avgDark_.begin(),
         std::bind1st(std::multiplies<float>(), 1/(float)numberOfProjections_));

   //read reference input values
   std::vector<unsigned short> referenceValues;
   readInput(pathReference_, referenceValues);

   computeAverage(referenceValues, avgReference_);
}

template<typename T>
auto Attenuation::computeAverage(const std::vector<T>& values,
      std::vector<float>&average) -> void {
   average.resize(numberOfProjections_ * numberOfDetectors_ * numberOfPlanes_);
   float factor = 1.0 / (float) numberOfRefFrames_;
   for (auto i = 0; i < numberOfRefFrames_; i++) {
      for (auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++) {
         for (auto index = 0; index < numberOfDetectors_ * numberOfProjections_;
               index++) {
            average[index + planeInd * numberOfDetectors_ * numberOfProjections_] +=
                  values[(i + planeInd) * numberOfProjections_
                        * numberOfDetectors_ + index] * factor;
         }
      }
   }
}

template<typename T>
auto Attenuation::readDarkInputFiles(std::string& path,
      std::vector<T>& values) -> void {
   if(path.back() != '/')
      path.append("/");
   std::ifstream input(path + "dark_192.168.100.fxc",
         std::ios::in | std::ios::binary);
   if (!input) {
      BOOST_LOG_TRIVIAL(error)<< "recoLib::cuda::Attenuation: Source file could not be loaded.";
      throw std::runtime_error("File could not be opened. Please check!");
   }
   //allocate memory in vector
   std::streampos fileSize;
   input.seekg(0, std::ios::end);
   fileSize = input.tellg();
   input.seekg(0, std::ios::beg);
   values.resize(numberOfDetectors_ * numberOfPlanes_);
   input.read((char*) &values[0],
         numberOfDetectors_ * numberOfPlanes_ * sizeof(T));
}

template<typename T>
auto Attenuation::readInput(std::string& path,
      std::vector<T>& values) -> void {
   std::vector<std::vector<T>> fileContents(numberOfDetectorModules_);
   Timer tmr1, tmr2;
   if(path.back() != '/')
      path.append("/");
   tmr1.start();
   tmr2.start();
#pragma omp parallel for default(shared) num_threads(9)
   for (auto i = 1; i <= numberOfDetectorModules_; i++) {
      std::vector<T> content;
      //TODO: make filename and ending configurable
      std::ifstream input(
            path + "ref_empty_tomograph_repaired_DetModNr_" + std::to_string(i)
                  + ".fx", std::ios::in | std::ios::binary);
      if (!input) {
         BOOST_LOG_TRIVIAL(error)<< "recoLib::cuda::Attenuation: Source file could not be loaded.";
         throw std::runtime_error("File could not be opened. Please check!");
      }
      //allocate memory in vector
      std::streampos fileSize;
      input.seekg(0, std::ios::end);
      fileSize = input.tellg();
      input.seekg(0, std::ios::beg);
      content.resize(fileSize / sizeof(T));
      input.read((char*) &content[0], fileSize);
      fileContents[i - 1] = content;
   }
   tmr2.stop();
   int numberOfDetPerModule = numberOfDetectors_ / numberOfDetectorModules_;
   values.resize(fileContents[0].size() * numberOfDetectorModules_);
   for (auto i = 0; i < numberOfRefFrames_; i++) {
      for (auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++) {
         for (auto projInd = 0; projInd < numberOfProjections_; projInd++) {
            for (auto detModInd = 0; detModInd < numberOfDetectorModules_;
                  detModInd++) {
               unsigned int startIndex = projInd * numberOfDetPerModule
                     + i * numberOfDetPerModule * numberOfProjections_;
               unsigned int indexSorted = detModInd * numberOfDetPerModule
                     + projInd * numberOfDetectors_
                     + (planeInd + i * numberOfPlanes_) * numberOfDetectors_ * numberOfProjections_;
               std::copy(fileContents[detModInd].begin() + startIndex,
                     fileContents[detModInd].begin() + startIndex
                           + numberOfDetPerModule,
                     values.begin() + indexSorted);
            }
         }
      }
   }
   tmr1.stop();
   double totalFileSize = numberOfProjections_*numberOfDetectors_*numberOfPlanes_*numberOfRefFrames_*sizeof(unsigned short)/1024.0/1024.0;
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Reading and sorting reference input took " << tmr1.elapsed() << " s, " << totalFileSize/tmr2.elapsed() << " MByte/s.";
}

template<typename T>
auto Attenuation::relevantAreaMask(std::vector<T>& mask) -> void {
   unsigned int ya, yb, yc, yd, ye;
   unsigned int yMin, yMax;
   double lowerLimit = (lowerLimOffset_ + sourceOffset_) / 360.0;
   double upperLimit = (upperLimOffset_ + sourceOffset_) / 360.0;
   //fill whole mask with ones and mask out the unrelevant parts afterwards
   mask.resize(numberOfProjections_ * numberOfDetectors_);
   std::fill(mask.begin(), mask.end(), 1.0);

   ya = std::round(lowerLimit * numberOfProjections_);
   yb = ya;
   yc = std::round(upperLimit * numberOfProjections_);
   yd = yc;

   //slope of the straight
   double m = ((double)ya - (double)yd) / ((double)xa_ - (double)xd_);

   ye = std::round((double)yc + ((double)xe_ - (double)xc_) * m);

   for (unsigned int x = 0; x <= xa_; x++) {
      yMin = ya;
      yMax = std::round(ye + m * x);
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xa_; x <= xc_; x++) {
      yMin = std::round(ya + m * (x - xa_));
      yMax = std::round(ye + m * x);
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xc_; x <= xd_; x++) {
      yMin = std::round(ya + m * (x - xa_));
      yMax = yd;
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xb_; x <= xf_; x++) {
      yMin = yb;
      yMax = std::round(yb + m * (x - xb_));
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   std::fill(mask.begin(),
         mask.begin() + lowerLimit * numberOfDetectors_ * numberOfProjections_,
         0.0);
   std::fill(
         mask.begin() + upperLimit * numberOfProjections_ * numberOfDetectors_,
         mask.end(), 0.0);
}

auto Attenuation::readConfig(const std::string& configFile) -> bool {
   ConfigReader configReader = ConfigReader(
         configFile.data());
   int samplingRate, scanRate;
   if (configReader.lookupValue("numberOfFanDetectors", numberOfDetectors_)
         && configReader.lookupValue("numberOfDetectorModules", numberOfDetectorModules_)
         && configReader.lookupValue("numberOfReferenceFrames", numberOfRefFrames_)
         && configReader.lookupValue("darkInputPath", pathDark_)
         && configReader.lookupValue("referenceInputPath", pathReference_)
         && configReader.lookupValue("numberOfPlanes", numberOfPlanes_)
         && configReader.lookupValue("samplingRate", samplingRate)
         && configReader.lookupValue("scanRate", scanRate)
         && configReader.lookupValue("sourceOffset", sourceOffset_)
         && configReader.lookupValue("xa", xa_)
         && configReader.lookupValue("xb", xb_)
         && configReader.lookupValue("xc", xc_)
         && configReader.lookupValue("xd", xd_)
         && configReader.lookupValue("xe", xe_)
         && configReader.lookupValue("xf", xf_)
         && configReader.lookupValue("lowerLimOffset", lowerLimOffset_)
         && configReader.lookupValue("upperLimOffset", upperLimOffset_)
         && configReader.lookupValue("blockSize2D_attenuation", blockSize2D_)
         && configReader.lookupValue("memPoolSize_attenuation", memPoolSize_)) {
      numberOfProjections_ = samplingRate * 1000000 / scanRate;
      return EXIT_SUCCESS;
   }

   return EXIT_FAILURE;
}

__global__ void computeAttenuation(
      const unsigned short* __restrict__ sinogram_in,
      const float* __restrict__ mask, float* __restrict__ sinogram_out,
      const float* __restrict__ avgReference, const float* __restrict__ avgDark,
      const float temp, const int numberOfDetectors,
      const int numberOfProjections, const int planeId) {

   auto x = ddrf::cuda::getX();
   auto y = ddrf::cuda::getY();
   if (x >= numberOfDetectors || y >= numberOfProjections)
      return;

   auto sinoIndex = numberOfDetectors * y + x;

   float numerator = (float) (sinogram_in[sinoIndex])
         - avgDark[planeId * numberOfDetectors + x];

   float denominator = avgReference[planeId * numberOfDetectors * numberOfProjections + sinoIndex]
         - avgDark[planeId * numberOfDetectors + x];

   if (numerator < temp)
      numerator = temp;
   if (denominator < temp)
      denominator = temp;

   //comutes the attenuation and multiplies with mask for hiding the unrelevant region
   sinogram_out[sinoIndex] = -log(numerator / denominator) * mask[sinoIndex];

}

}
}
